#include "hip/hip_runtime.h"
#include "TimeWarpManager.h"

#include "MyExceptions.h"
#include "GLManager.h"

#include <thrust\reduce.h>
#include <thrust\functional.h>
#include <iostream>
#include <algorithm>
#include <boost\timer.hpp>

using namespace std;


TimeWarpManager::TimeWarpManager(u32 maxPlane, u32 maxSpheres)
	:MAX_PLANES(maxPlane), MAX_SPHERES(maxSpheres), MAX_CUBOIDS(0), 
	MAX_FIXCUBOIDS(0), planeCount(0), sphereCount(0), cuboidCount(0),
	MAX_LEAFINDICES(0), leafIndexCount(0), 
	MAX_CONNECTORS(0), connectorCount(0)
{
	hipSetDeviceFlags(hipDeviceMapHost);
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);


	throwCudaError(hipHostAlloc(&planes, MAX_PLANES*sizeof(Plane), 
		hipHostMallocMapped));
	throwCudaError(hipHostGetDevicePointer(&cuPlanes, planes, 0));

	throwCudaError(hipHostAlloc(&spheres, MAX_SPHERES*sizeof(Sphere), 
		hipHostMallocMapped));
	throwCudaError(hipHostGetDevicePointer(&cuSpheres, spheres, 0));

	throwCudaError(hipHostAlloc(&outputQs, MAX_SPHERES*sizeof(Queue<Message, QL>), 
		hipHostMallocMapped));
	throwCudaError(hipHostGetDevicePointer(&cuOutputQs, outputQs, 0));

	throwCudaError(hipHostAlloc(&stateQs, MAX_SPHERES*sizeof(Queue<Sphere, QL>), 
		hipHostMallocMapped));
	throwCudaError(hipHostGetDevicePointer(&cuStateQs, stateQs, 0));

	throwCudaError(hipHostAlloc(&lvts, MAX_SPHERES*sizeof(f32), 
		hipHostMallocMapped));
	throwCudaError(hipHostGetDevicePointer(&cuLvts, lvts, 0));
	
	throwCudaError(hipHostAlloc(&mailboxes, MAX_SPHERES*sizeof(Queue<Message, QL>), 
		hipHostMallocMapped));
	throwCudaError(hipHostGetDevicePointer(&cuMailboxes, mailboxes, 0));

	throwCudaError(hipHostAlloc(&pendings, MAX_SPHERES*sizeof(Sphere), 
		hipHostMallocMapped));
	throwCudaError(hipHostGetDevicePointer(&cuPendings, pendings, 0));
	
	throwCudaError(hipHostAlloc(&inputQs, MAX_SPHERES*sizeof(Heap<Message, QL>), 
		hipHostMallocMapped));
	throwCudaError(hipHostGetDevicePointer(&cuInputQs, inputQs, 0));

}


TimeWarpManager::~TimeWarpManager(){
	hipHostFree(planes);
	hipHostFree(spheres);
	hipHostFree(outputQs);
	hipHostFree(stateQs);
	hipHostFree(lvts);
	hipHostFree(mailboxes);
	hipHostFree(pendings);
	hipHostFree(inputQs);
}

u32 TimeWarpManager::addSphere(int n){
	u32 begin = sphereCount;
	sphereCount += n;
	if(sphereCount >= MAX_SPHERES){
		sphereCount = MAX_SPHERES;
	}

	for(u32 i = begin; i < sphereCount; i++){
		spheres[i] = Sphere();
		GLManager::instance().add(spheres[i]);
	}
	return begin;
}

	
u32 TimeWarpManager::addPlane(int n){
	u32 begin = planeCount;
	planeCount += n;
	if(planeCount >= MAX_PLANES){
		planeCount = MAX_PLANES;
	}

	for(u32 i = begin; i < planeCount; i++){
		planes[i] = Plane();
		GLManager::instance().add(planes[i]);
	}
	return begin;
}


void TimeWarpManager::calculateTime(f64 dt, f64 div){
	boost::timer timer;

	dt /= div;

	cpToStateQs<<<sphereCount/BSIZE+1, BSIZE>>>(cuSpheres, cuPendings, cuStateQs, sphereCount);
	throwCudaError(hipDeviceSynchronize());

	f64 gvt = 0;

	while(gvt < dt){
		//initiale Kollisionen bestimmen und entsprechende Nachrichten verschicken
		detectCollisions<<<sphereCount/BSIZE+1, BSIZE>>>(cuPlanes, planeCount, cuMailboxes, cuPendings, cuOutputQs, cuStateQs, sphereCount, dt);
		throwCudaError(hipDeviceSynchronize());

		/*if(!stateQs[0].back().r)
			cout << "bullshit" << endl;*/

		//Nachrichten in die inputQs stecken
		receiveFromMailboxes<<<sphereCount/BSIZE+1, BSIZE>>>(cuInputQs, cuMailboxes, sphereCount);
		throwCudaError(hipDeviceSynchronize());
		removeFromMailboxes<<<sphereCount/BSIZE+1, BSIZE>>>(cuMailboxes, sphereCount);
		throwCudaError(hipDeviceSynchronize());

		/*if(!stateQs[0].back().r)
			cout << "bullshit" << endl;*/

		//inputQs abarbeiten
		handleNextMessages<<<sphereCount/BSIZE+1, BSIZE>>>(cuStateQs, cuInputQs, cuOutputQs, cuPendings, cuMailboxes, sphereCount);
		throwCudaError(hipDeviceSynchronize());

		/*if(!stateQs[0].back().r)
			cout << "bullshit" << endl;*/

		//verschickte antimesseges in die inputQs stecken
		receiveFromMailboxes<<<sphereCount/BSIZE+1, BSIZE>>>(cuInputQs, cuMailboxes, sphereCount);
		throwCudaError(hipDeviceSynchronize());
		removeFromMailboxes<<<sphereCount/BSIZE+1, BSIZE>>>(cuMailboxes, sphereCount);
		throwCudaError(hipDeviceSynchronize());

		/*if(!stateQs[0].back().r)
			cout << "bullshit" << endl;*/

		//rollbacks etc durchfuehren
		handleNextMessages<<<sphereCount/BSIZE+1, BSIZE>>>(cuStateQs, cuInputQs, cuOutputQs, cuPendings, cuMailboxes, sphereCount);
		throwCudaError(hipDeviceSynchronize());

		/*if(!stateQs[0].back().r)
			cout << "bullshit" << endl;*/

		//mailbox leeren
		receiveFromMailboxes<<<sphereCount/BSIZE+1, BSIZE>>>(cuInputQs, cuMailboxes, sphereCount);
		throwCudaError(hipDeviceSynchronize());
		removeFromMailboxes<<<sphereCount/BSIZE+1, BSIZE>>>(cuMailboxes, sphereCount);
		throwCudaError(hipDeviceSynchronize());

		/*if(!stateQs[0].back().r)
			cout << "bullshit" << endl;*/

		//neue gvt berechnen
		calculateLVT<<<sphereCount/BSIZE+1, BSIZE>>>(cuInputQs, cuStateQs, cuLvts, sphereCount);
		throwCudaError(hipDeviceSynchronize());
		gvt = thrust::reduce(lvts, lvts+sphereCount, 1000000., thrust::min<f64>);

		/*if(!stateQs[0].back().r)
			cout << "bullshit" << endl;*/
		
		//alte Sachen loeschen
		deleteOlderThanGVT<<<sphereCount/BSIZE+1, BSIZE>>>(cuOutputQs, cuStateQs, sphereCount, gvt);
		throwCudaError(hipDeviceSynchronize());

		/*if(!stateQs[0].back().r)
			cout << "bullshit" << endl;*/

		//cout << "max Q length: " << max_element(stateQs, stateQs+sphereCount)->length() << endl;;
	}
	/*if(gvt > dt)
		cout << "gvt > dt" << endl;*/
	
	cpFromStateQs<<<sphereCount/BSIZE+1, BSIZE>>>(cuSpheres, cuStateQs, sphereCount);
	throwCudaError(hipDeviceSynchronize());

	accelerate<<<sphereCount/BSIZE+1, BSIZE>>>(cuSpheres, sphereCount, gvt);
	throwCudaError(hipDeviceSynchronize());

	cout << fixed << "Berechnungszeit fuer einen Frame: " << (int)(timer.elapsed()*1000) << "ms\t\r";
	//cout << "Kugel 0 pos: ("<<spheres[0].x[0]<<", "<<spheres[0].x[1]<<", "<<spheres[0].x[2]<<")\r";
}