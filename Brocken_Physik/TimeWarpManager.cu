#include "hip/hip_runtime.h"
#include "TimeWarpManager.h"

#include "MyExceptions.h"
#include "GLManager.h"

#include <thrust\reduce.h>
#include <thrust\functional.h>


TimeWarpManager::TimeWarpManager(u32 maxPlane, u32 maxSpheres)
	:MAX_PLANES(maxPlane), MAX_SPHERES(maxSpheres), MAX_CUBOIDS(0), 
	MAX_FIXCUBOIDS(0), planeCount(0), sphereCount(0), cuboidCount(0),
	MAX_LEAFINDICES(0), leafIndexCount(0), 
	MAX_CONNECTORS(0), connectorCount(0)
{
	hipSetDeviceFlags(hipDeviceMapHost);
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);


	throwCudaError(hipHostAlloc(&planes, MAX_PLANES*sizeof(Plane), 
		hipHostMallocMapped));
	throwCudaError(hipHostGetDevicePointer(&cuPlanes, planes, 0));

	throwCudaError(hipHostAlloc(&spheres, MAX_SPHERES*sizeof(Sphere), 
		hipHostMallocMapped));
	throwCudaError(hipHostGetDevicePointer(&cuSpheres, spheres, 0));

	throwCudaError(hipHostAlloc(&outputQs, MAX_SPHERES*sizeof(Queue<Message, QL>), 
		hipHostMallocMapped));
	throwCudaError(hipHostGetDevicePointer(&cuOutputQs, outputQs, 0));

	throwCudaError(hipHostAlloc(&stateQs, MAX_SPHERES*sizeof(Queue<Sphere, QL>), 
		hipHostMallocMapped));
	throwCudaError(hipHostGetDevicePointer(&cuStateQs, stateQs, 0));

	throwCudaError(hipHostAlloc(&lvts, MAX_SPHERES*sizeof(f32), 
		hipHostMallocMapped));
	throwCudaError(hipHostGetDevicePointer(&cuLvts, lvts, 0));
	
	throwCudaError(hipHostAlloc(&mailboxes, MAX_SPHERES*sizeof(Queue<Message, QL>), 
		hipHostMallocMapped));
	throwCudaError(hipHostGetDevicePointer(&cuMailboxes, mailboxes, 0));

	throwCudaError(hipHostAlloc(&pendings, MAX_SPHERES*sizeof(Sphere), 
		hipHostMallocMapped));
	throwCudaError(hipHostGetDevicePointer(&cuPendings, pendings, 0));
	
	throwCudaError(hipHostAlloc(&inputQs, MAX_SPHERES*sizeof(Heap<Message, QL>), 
		hipHostMallocMapped));
	throwCudaError(hipHostGetDevicePointer(&cuInputQs, inputQs, 0));

}


TimeWarpManager::~TimeWarpManager(){
	hipHostFree(planes);
	hipHostFree(spheres);
	hipHostFree(outputQs);
	hipHostFree(stateQs);
	hipHostFree(lvts);
	hipHostFree(mailboxes);
	hipHostFree(pendings);
	hipHostFree(inputQs);
}

u32 TimeWarpManager::addSphere(int n){
	u32 begin = sphereCount;
	sphereCount += n;
	if(sphereCount >= MAX_SPHERES){
		sphereCount = MAX_SPHERES;
	}

	for(u32 i = begin; i < sphereCount; i++){
		spheres[i] = Sphere();
		GLManager::instance().add(spheres[i]);
	}
	return begin;
}

	
u32 TimeWarpManager::addPlane(int n){
	u32 begin = planeCount;
	planeCount += n;
	if(planeCount >= MAX_PLANES){
		planeCount = MAX_PLANES;
	}

	for(u32 i = begin; i < planeCount; i++){
		planes[i] = Plane();
		GLManager::instance().add(planes[i]);
	}
	return begin;
}


void TimeWarpManager::calculateTime(f32 dt, f32 div){
	dt /= div;

	cpToStateQs<<<sphereCount/BSIZE+1, BSIZE>>>(cuSpheres, cuPendings, cuStateQs, sphereCount);
	throwCudaError(hipDeviceSynchronize());

	f32 gvt = 0;

	while(gvt < dt){
		//initiale Kollisionen bestimmen und entsprechende Nachrichten verschicken
		detectCollisions<<<sphereCount/BSIZE+1, BSIZE>>>(cuPlanes, planeCount, cuMailboxes, cuPendings, cuOutputQs, cuStateQs, sphereCount, dt);
		throwCudaError(hipDeviceSynchronize());

		//Nachrichten in die inputQs stecken
		receiveFromMailboxes<<<sphereCount/BSIZE+1, BSIZE>>>(cuInputQs, cuMailboxes, sphereCount);
		throwCudaError(hipDeviceSynchronize());
		removeFromMailboxes<<<sphereCount/BSIZE+1, BSIZE>>>(cuMailboxes, sphereCount);
		throwCudaError(hipDeviceSynchronize());

		//inputQs abarbeiten
		handleNextMessages<<<sphereCount/BSIZE+1, BSIZE>>>(cuStateQs, cuInputQs, cuOutputQs, cuPendings, cuMailboxes, sphereCount);
		throwCudaError(hipDeviceSynchronize());

		//verschickte antimesseges in die inputQs stecken
		receiveFromMailboxes<<<sphereCount/BSIZE+1, BSIZE>>>(cuInputQs, cuMailboxes, sphereCount);
		throwCudaError(hipDeviceSynchronize());
		removeFromMailboxes<<<sphereCount/BSIZE+1, BSIZE>>>(cuMailboxes, sphereCount);
		throwCudaError(hipDeviceSynchronize());

		//rollbacks etc durchfuehren
		handleNextMessages<<<sphereCount/BSIZE+1, BSIZE>>>(cuStateQs, cuInputQs, cuOutputQs, cuPendings, cuMailboxes, sphereCount);
		throwCudaError(hipDeviceSynchronize());

		//mailbox leeren
		receiveFromMailboxes<<<sphereCount/BSIZE+1, BSIZE>>>(cuInputQs, cuMailboxes, sphereCount);
		throwCudaError(hipDeviceSynchronize());
		removeFromMailboxes<<<sphereCount/BSIZE+1, BSIZE>>>(cuMailboxes, sphereCount);
		throwCudaError(hipDeviceSynchronize());

		//neue gvt berechnen
		calculateLVT<<<sphereCount/BSIZE+1, BSIZE>>>(cuInputQs, cuStateQs, cuLvts, sphereCount);
		throwCudaError(hipDeviceSynchronize());
		gvt = thrust::reduce(lvts, lvts+sphereCount, 1000000.f, thrust::min<f32>);
		
		//alte Sachen loeschen
		deleteOlderThanGVT<<<sphereCount/BSIZE+1, BSIZE>>>(cuOutputQs, cuStateQs, sphereCount, gvt);
		throwCudaError(hipDeviceSynchronize());
	}
	
	cpFromStateQs<<<sphereCount/BSIZE+1, BSIZE>>>(cuSpheres, cuStateQs, sphereCount, gvt);
	throwCudaError(hipDeviceSynchronize());
}